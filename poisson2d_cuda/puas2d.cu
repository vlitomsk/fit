// orig http://ssd.sscc.ru/sites/default/files/content/attach/343/puassonv3dv2.pdf
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define sqr(x) ((x)*(x))
#define cudaCheck \
    do { \
        hipError_t err=hipGetLastError(); \
        if( err != hipSuccess ) { \
            printf(" hipError_t = '%s' \n in '%s' %d \n", \
                    hipGetErrorString(err), \
                    __FILE__, __LINE__ ); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define tid threadIdx
#define bid blockIdx
#define bdim blockDim

/* * * * * */
/* TASK PARAMS */
/* * * * * */

#define a 1.0f
/* Размеры сердцевины */
#define in 16
#define jn 16

#ifndef bsx
#define bsx 4
#endif

#ifndef bsy
#define bsy 4
#endif

const float X = 2.0f, Y = 2.0f;
const float hx = X/(in+1), hy = Y/(jn+1);
const float owx_rev = 1.0f/(hx*hx), owy_rev =1.0f/( hy*hy);
const float c_rev = 1.0f/(2.0f*owx_rev + 2.0f*owy_rev + a);

const float eps = 1e-5;

/* * * * * */
/* END TASK PARAMS */
/* * * * * */

void pmat(float *m, int w, int h) {
  for (int i = 0; i < h; ++i) {
    for (int j = 0; j < w; ++j) {
      printf("%.2f ", m[i*w + j]);
    }
    printf("\n");
  }
}


float Fresh_host(float x, float y) {
    return x+y;
}

__device__ float Fresh(float x, float y) {
    return x+y;
}

__device__ float Rho(float x, float y) {
    return -a*(x+y);
}

float Rho_host(float x, float y) {
    return -a*(x+y);
}

__device__ int lessthan_eps = 1; // 1 true, 0 false

#define cbsx (bsx+2)
#define cbsy (bsy+2)
#define fsx (in+2)
#define fsy (jn+2)
#define cbar(i,j) subcube[(j)*cbsx + (i)]
#define far(layer,i,j) fdev[fsx*fsy*(layer) + (j)*fsx + (i)]
#define far0(i,j) far(l0,(i),(j))
#define far1(i,j) far(l1,(i),(j))
#define far_h(layer,i,j) f_host[(layer)*fsx*fsy + (j)*fsx + (i)]

__global__ void initmat(float * fdev) {
    const int i = 1 + tid.x + bsx * bid.x,
              j = 1 + tid.y + bsy * bid.y;

    far(0,i,j) = far(1,i,j) = 0;

    if (i == 1) {
        far(0,0,j) = far(1,0,j) = Fresh(0*hx, j*hy);
        far(0,fsx-1,j) = far(1,fsx-1,j) = Fresh((fsx-1)*hx, j*hy);
    }

    if (j == 1) {
        far(0,i,0) = far(1,i,0) = Fresh(i*hx, 0*hy);
        far(0,i,fsy-1) = far(1,i,fsy-1) = Fresh(i*hx, (fsy-1)*hy);
    }
}

__global__ void dostep(float * fdev, int l0, int l1, bool cpsymbol) {
    __shared__ float subcube[cbsx*cbsy];
    __shared__ int lteps ;
    lteps = 1;
    // fij -- индекс потока в вычислительной сетке размером (in+2)*(jn+2) = fsx * fsy
    // cbij -- индекс потока в shared-области размером (bsx+2)*(bsy+2) = cbsx * cbsy
    const int cbi = 1 + tid.x,
              cbj = 1 + tid.y;
    const int fi = cbi + bsx * bid.x,
              fj = cbj + bsy * bid.y;

    cbar(cbi, cbj) = far0(fi, fj);
    const float old_val = cbar(cbi,cbj);

    if (tid.x <= 1) {
        int idx = tid.x * (bsx+1);
        cbar(idx,cbj) = far0(fi + idx - cbi, fj);
    }

    if (tid.y  <= 1) {
        int idx = tid.y * (bsy+1);
        cbar(cbi, idx) = far0(fi, fj + idx - cbj);
    }

    __syncthreads();
    const float Fival = (cbar(cbi+1, cbj) + cbar(cbi-1, cbj)) * owx_rev;
    const float Fjval = (cbar(cbi, cbj+1) + cbar(cbi, cbj-1)) * owy_rev;
    const float new_val = (Fival + Fjval - Rho(fi*hx, fj*hy)) * c_rev;
    far1(fi, fj) = new_val;

    if (cpsymbol) {
        if ( lteps && fabs(new_val - old_val) > eps )  {
            lteps = 0;
        }
        __syncthreads();
        if (tid.x == 0 && tid.y ==0 && !lteps) {
            lessthan_eps = 0;
        }
    }
}

int main(int argc, char ** argv) {
    hipSetDevice(0); cudaCheck;
    hipEvent_t ev_start, ev_end;
    hipEventCreate(&ev_start); cudaCheck;
    hipEventCreate(&ev_end); cudaCheck;

    /* * * * */

    int bytesz = 2 *fsx * fsy * sizeof(float);
    float * f_host = (float*) malloc(bytesz);
    float * f_dev;
    if (!f_host) {
        perror("malloc");
        return EXIT_FAILURE;
    }

    hipMalloc(&f_dev, bytesz); cudaCheck;


    /** DO ALL WORK HERE **/

    const bool profena = false;
    const int epsCheckFreq = 10;
    int lt_eps, iter = 0;

    dim3 gdim = dim3(in/bsx, jn/bsy, 1);
    dim3 bdim = dim3(bsx, bsy, 1);

    initmat <<< gdim, bdim >>> ( f_dev ); cudaCheck;
    hipDeviceSynchronize(); cudaCheck;

    if (profena) {
        hipProfilerStart(); cudaCheck;
    }

    hipEventRecord(ev_start, 0); cudaCheck;
    int *cuda_lteps;
    hipGetSymbolAddress((void**)&cuda_lteps, HIP_SYMBOL("lessthan_eps"));
    do {
        bool cpsymbol = iter % epsCheckFreq == 0;
        if (cpsymbol) {
            lt_eps = 1;
            hipMemcpy(cuda_lteps, &lt_eps, sizeof(int), hipMemcpyHostToDevice); cudaCheck;
        }
        dostep <<< gdim, bdim >>> ( f_dev, iter&1, (iter&1)^1, cpsymbol ); cudaCheck;
        hipDeviceSynchronize(); cudaCheck;
        if (cpsymbol) {
            hipMemcpy(&lt_eps, cuda_lteps, sizeof(int), hipMemcpyDeviceToHost); cudaCheck;
        }
        ++iter;
    } while (iter < 1000 && lt_eps == 0);

    hipEventRecord(ev_end, 0); cudaCheck;

    if (profena) {
        hipProfilerStop(); cudaCheck;
    }

    /** copy result to HOST and check */

    hipMemcpy(f_host, f_dev, bytesz, hipMemcpyDeviceToHost); cudaCheck;

    if( fsx<=32 && fsy<=32 ) {
        pmat(f_host, fsx,fsy);
        printf("\n\n");
    }

    float maxerr = 0.0f;
    int mi, mj;
    for (int i = 1; i < in + 1; ++i) {
        for (int j = 1; j < jn + 1; ++j) {
            float fl = fabs(far_h(0,i,j) - Fresh_host(i*hx, j*hy));
            if (fl > maxerr){
                maxerr = fl;
                mi = i;
                mj = j;
            }
        }
    }
    printf("iters: %3d; <%3d %3d> : maxerr: %f fresh = %f\n", iter, mi,mj, maxerr, Fresh_host( mi*hx, mj*hy ) );

    /* * * * */
    float time_took;
    hipEventElapsedTime(&time_took, ev_start, ev_end); cudaCheck;
    printf("Time: %f\n", time_took);

    free(f_host);
    hipFree(f_dev); cudaCheck;


    return EXIT_SUCCESS;
}

